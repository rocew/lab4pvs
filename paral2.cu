#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

int safe_input(const char* prompt) {
    int value;
    while (1) {
        printf("%s", prompt);
        if (scanf("%d", &value) == 1 && value > 0) {
            break;
        }
        printf("Некорректный ввод. Пожалуйста, введите положительное целое число.\n");
        // Очистка буфера ввода
        while (getchar() != '\n');
    }
    return value;
}

__global__ void bitonic_sort_step(int* data, int j, int k, int n) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n) return;

    unsigned int ixj = i ^ j;

    if (ixj > i) {
        if ((i & k) == 0) {
            if (data[i] > data[ixj]) {
                int temp = data[i];
                data[i] = data[ixj];
                data[ixj] = temp;
            }
        }
        else {
            if (data[i] < data[ixj]) {
                int temp = data[i];
                data[i] = data[ixj];
                data[ixj] = temp;
            }
        }
    }
}

void generate_random_array(int* arr, int size) {
    for (int i = 0; i < size; i++)
        arr[i] = rand() % 1000000;
}

int main() {
    int SIZE = safe_input("Введите размер массива (рекомендуется степень двойки): ");
    int RUNS = safe_input("Введите количество запусков: ");

    int* h_array = (int*)malloc(SIZE * sizeof(int));
    int* d_array;

    int threads_list[] = { 4, 8, 16 };

    hipMalloc((void**)&d_array, SIZE * sizeof(int));

    for (int t = 0; t < sizeof(threads_list) / sizeof(threads_list[0]); t++) {
        int threads = threads_list[t];
        int blocks = (SIZE + threads - 1) / threads;

        float* times = (float*)malloc(RUNS * sizeof(float));
        float total_time = 0.0f;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        for (int r = 0; r < RUNS; r++) {
            generate_random_array(h_array, SIZE);
            hipMemcpy(d_array, h_array, SIZE * sizeof(int), hipMemcpyHostToDevice);

            hipEventRecord(start);

            for (int k = 2; k <= SIZE; k <<= 1) {
                for (int j = k >> 1; j > 0; j >>= 1) {
                    bitonic_sort_step << <blocks, threads >> > (d_array, j, k, SIZE);
                    hipDeviceSynchronize();
                }
            }

            hipEventRecord(stop);
            hipEventSynchronize(stop);

            float ms = 0;
            hipEventElapsedTime(&ms, start, stop);
            times[r] = ms / 1000.0f;  // перевод в секунды
            total_time += times[r];
        }

        printf("\nThreads: %d\n", threads);
        printf("Array size: %d\n", SIZE);

        printf("First 5 runs:\n");
        for (int i = 0; i < 5 && i < RUNS; i++) {
            printf("Run %d: %.6f sec\n", i + 1, times[i]);
        }

        printf("\nLast 5 runs:\n");
        for (int i = (RUNS > 5) ? RUNS - 5 : 0; i < RUNS; i++) {
            printf("Run %d: %.6f sec\n", i + 1, times[i]);
        }

        printf("\nAverage time: %.6f sec\n", total_time / RUNS);

        free(times);
        hipDeviceSynchronize();
    }

    hipFree(d_array);
    free(h_array);
    return 0;
}
