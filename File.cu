#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define runs 100
__global__ void sum_array_kernel(float* array, float* result, int n) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        atomicAdd(result, array[i]);
    }
}

int main() {
    int n = 10000000; // ����� 100000 ���������
    float* array = (float*)malloc(n * sizeof(float));
    float sum = 0.0f;

    // ������������� �������
    for (int i = 0; i < n; i++) {
        array[i] = rand() / (float)RAND_MAX;
    }

    // ��������� ������ �� GPU
    float* d_array;
    float* d_sum;
    hipMalloc((void**)&d_array, n * sizeof(float));
    hipMalloc((void**)&d_sum, sizeof(float));

    // ����������� ������ �� GPU
    hipMemcpy(d_array, array, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemset(d_sum, 0, sizeof(float)); // ������������� ����� �� GPU

    double total_time = 0;
    double times[runs];
    int threads_list[] = { 4,8,16 }; // ������ �������� ���������� �������

    for (int t = 0; t < sizeof(threads_list) / sizeof(threads_list[0]); t++) {
        int threads = threads_list[t];
        int blocks = (n + threads - 1) / threads;

        hipEvent_t start, stop;
        hipEventCreate(&start);
        hipEventCreate(&stop);

        total_time = 0;
        for (int r = 0; r < runs; r++) {
            hipMemset(d_sum, 0, sizeof(float)); // ����� ����� ����� ������ ��������

            hipEventRecord(start);
            sum_array_kernel << <blocks, threads >> > (d_array, d_sum, n);
            hipEventRecord(stop);
            hipEventSynchronize(stop);

            float ms = 0;
            hipEventElapsedTime(&ms, start, stop);
            times[r] = ms / 1000.0; // ��������� � �������
            total_time += times[r];
        }

        // �������� ��������� ������� �� CPU ��� ��������
        hipMemcpy(&sum, d_sum, sizeof(float), hipMemcpyDeviceToHost);

        printf("\nThreads: %d\n", threads);
        printf("Array size: %d\n", n);

        printf("First 5 runs:\n");
        for (int i = 0; i < 5; i++) {
            printf("Run %d: %.6f sec\n", i + 1, times[i]);
        }

        printf("\nLast 5 runs:\n");
        for (int i = runs - 5; i < runs; i++) {
            printf("Run %d: %.6f sec\n", i + 1, times[i]);
        }

        printf("\nAverage time: %.6f sec\n", total_time / runs);
    }

    // ������������ ������
    hipFree(d_array);
    hipFree(d_sum);
    free(array);

    return 0;
}